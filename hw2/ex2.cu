#include "hip/hip_runtime.h"
#include "ex2.h"
#include <cuda/atomic>

#define STREAM_THREADS 1024
#define COMMON_SIZE 256
#define INTERPOLATE_MEM 1024
#define REGS_PER_THREAD 32

__device__ void prefix_sum(int arr[], int arr_size) {
    // TODO complete according to hw1
    int tid = threadIdx.x;
    int num_threads = blockDim.x;
    extern __shared__ int temp[];
    for (int stride = 1; stride < arr_size; stride *= 2) {
        for (int i = tid; i < arr_size; i += num_threads) {
            if (i >= stride) {
                temp[i] = arr[i - stride];
            }
        }
        __syncthreads();
        for (int i = tid; i < arr_size; i += num_threads) {
            if (i >= stride) {
                arr[i] += temp[i];
            }
        }
        __syncthreads();
    }
    return;
}

/**
 * Perform interpolation on a single image
 *
 * @param maps 3D array ([TILES_COUNT][TILES_COUNT][256]) of    
 *             the tiles’ maps, in global memory.
 * @param in_img single input image, in global memory.
 * @param out_img single output buffer, in global memory.
 */
__device__
 void interpolate_device(uchar* maps ,uchar *in_img, uchar* out_img);

__device__
void process_image(uchar *in, uchar *out, uchar* maps) {
    // TODO complete according to hw1
    int tid = threadIdx.x;
    int block_idx = blockIdx.x;
    int block_size = blockDim.x;
    __shared__ int sharedHistogram[COMMON_SIZE];
    uchar* curr_in = in + block_idx * IMG_HEIGHT * IMG_WIDTH;
    uchar* curr_out = out + block_idx * IMG_HEIGHT * IMG_WIDTH;
    uchar* curr_maps = &maps[block_idx * TILE_COUNT * TILE_COUNT * COMMON_SIZE];
    // do one tile at a time
    for (int row_tile_n = 0; row_tile_n < TILE_COUNT; row_tile_n++) {
        for (int col_tile_n = 0; col_tile_n < TILE_COUNT; col_tile_n++) {
            // Initialize sharedHistogram/reset everything to zero
            for (int k = tid; k < COMMON_SIZE; k += block_size) {
                sharedHistogram[k] = 0;
            }
            __syncthreads();

            // // Debug print for shared memory initialization
            // if (tid == 0) {
            //     printf("Shared Histogram Initialized:\n");
            //     for (int i = 0; i < COMMON_SIZE; i++) {
            //         printf("%d: %d\t", i, sharedHistogram[i]);
            //     }
            //     printf("\n");
            // }
            // __syncthreads();

            // Fill histogram
            for (int i = tid; i < TILE_WIDTH * TILE_WIDTH; i += block_size) {
                int tile_col = i % TILE_WIDTH;
                int tile_row = i / TILE_WIDTH;
                int y = TILE_WIDTH * row_tile_n + tile_row;
                int x = TILE_WIDTH * col_tile_n + tile_col;
                uchar* row = curr_in + y * IMG_WIDTH;
                atomicAdd(&sharedHistogram[row[x]], 1);
            }
            __syncthreads(); // Ensure all atomic adds are done

            // // Debug print for histogram values
            // if (tid == 0) {
            //     printf("Shared Histogram Filled:\n");
            //     for (int i = 0; i < COMMON_SIZE; i++) {
            //         printf("%d: %d\t", i, sharedHistogram[i]);
            //     }
            //     printf("\n");
            // }
            // __syncthreads();

            // Prefix sum on sharedHistogram
            prefix_sum(sharedHistogram, COMMON_SIZE);
            __syncthreads(); // Ensure prefix sum is completed

            // // Debug print for prefix sum values
            // if (tid == 0) {
            //     printf("Shared Histogram After Prefix Sum:\n");
            //     for (int i = 0; i < COMMON_SIZE; i++) {
            //         printf("%d: %d\t", i, sharedHistogram[i]);
            //     }
            //     printf("\n");
            // }
            // __syncthreads();

            // Get correct maps entry
            uchar* map = &curr_maps[row_tile_n * TILE_COUNT * COMMON_SIZE + col_tile_n * COMMON_SIZE];
            // Create new map values
            for (int k = tid; k < COMMON_SIZE; k += block_size) {
                map[k] = (float(sharedHistogram[k]) * 255) / (TILE_WIDTH * TILE_WIDTH);
            }
            __syncthreads();
        }
    }

    __syncthreads();
    interpolate_device(curr_maps, curr_in, curr_out);
    return;
}

__global__
void process_image_kernel(uchar *in, uchar *out, uchar* maps){
    process_image(in, out, maps);
}

class streams_server : public image_processing_server
{
private:
    // TODO define stream server context (memory buffers, streams, etc...)
    hipStream_t streams[STREAM_COUNT];
    uchar* maps;
    uchar* in_image;
    uchar* out_image;
    int image_id[STREAM_COUNT];

public:
    streams_server()
    {
        // TODO initialize context (memory buffers, streams, etc...)
        for (int i = 0; i < STREAM_COUNT; i++)
        {
            CUDA_CHECK(hipStreamCreate(&streams[i]));
        }
        CUDA_CHECK(hipHostAlloc(&maps, sizeof(uchar) * STREAM_COUNT * TILE_COUNT * TILE_COUNT * COMMON_SIZE, 0));
        CUDA_CHECK(hipHostAlloc(&in_image, sizeof(uchar) * STREAM_COUNT * IMG_HEIGHT * IMG_WIDTH, 0));
        CUDA_CHECK(hipHostAlloc(&out_image, sizeof(uchar) * STREAM_COUNT * IMG_HEIGHT * IMG_WIDTH, 0));
    }

    ~streams_server() override
    {
        // TODO free resources allocated in constructor
        for (int i = 0; i < STREAM_COUNT; i++)
        {
            CUDA_CHECK(hipStreamDestroy(streams[i]));
        }
        CUDA_CHECK(hipHostFree(maps));
        CUDA_CHECK(hipHostFree(in_image));
        CUDA_CHECK(hipHostFree(out_image));
    }

    bool enqueue(int img_id, uchar *img_in, uchar *img_out) override
    {
        // TODO place memory transfers and kernel invocation in streams if possible.
        for (int i = 0; i < STREAM_COUNT; i++)
        {
            if (image_id[i] == -1)
            {
                image_id[i] = img_id;
                uchar* maps_d = this->maps + i * TILE_COUNT * TILE_COUNT * COMMON_SIZE;
                uchar* in_img_d = this->in_image + i * IMG_HEIGHT * IMG_WIDTH;
                uchar* out_img_d = this->out_image + i * IMG_HEIGHT * IMG_WIDTH;
                CUDA_CHECK(hipMemcpyAsync(in_img_d, img_in, sizeof(uchar) * IMG_HEIGHT * IMG_WIDTH, 
                                            hipMemcpyHostToDevice, streams[i]));
                process_image_kernel<<<1, STREAM_THREADS, sizeof(int) * COMMON_SIZE, streams[i]>>>( in_img_d,
                                                                                                    out_img_d, 
                                                                                                    maps_d);
                CUDA_CHECK(hipMemcpyAsync(img_out, out_img_d, sizeof(uchar) * IMG_HEIGHT * IMG_WIDTH, 
                                            hipMemcpyDeviceToHost, streams[i]));
                return true;
            }
        }
        return false;
    }

    bool dequeue(int *img_id) override
    {
        return false;

        // TODO query (don't block) streams for any completed requests.
        for (int i = 0; i < STREAM_COUNT; i++)
        {
            hipError_t status = hipStreamQuery(streams[i]); // TODO query diffrent stream each iteration
            switch (status) {
            case hipSuccess:
                // TODO return the img_id of the request that was completed.
                *img_id = image_id[i];
                image_id[i] = -1;
                return true;
            case hipErrorNotReady:
                return false;
            default:
                CUDA_CHECK(status);
                return false;
            }
        }
    }
};

std::unique_ptr<image_processing_server> create_streams_server()
{
    return std::make_unique<streams_server>();
}

// TODO implement a Test and Test and Set lock
class TATASLock
{
private:
    // TODO define lock context
    cuda::atomic<int, cuda::thread_scope_device> gpu_lock_state;
    std::atomic<int> cpu_lock_state;

public:
    __host__ __device__ TATASLock() : gpu_lock_state(0), cpu_lock_state(0) {}

    // GPU lock
    __device__ void gpu_lock() {
        while (true) {
            // First check if the lock appears to be free
            if (gpu_lock_state.load(cuda::memory_order_acquire) == 0) {
                // Attempt to acquire the lock
                if (gpu_lock_state.exchange(1, cuda::memory_order_acquire) == 0) {
                    return; // Successfully acquired the lock
                }
            }
            // Spin-wait (busy-wait) until the lock is free
        }
    }

    // GPU unlock
    __device__ void gpu_unlock() {
        gpu_lock_state.store(0, cuda::memory_order_release);
    }

    // CPU lock
    void cpu_lock() {
        while (true) {
            // First check if the lock appears to be free
            if (cpu_lock_state.load(std::memory_order_acquire) == 0) {
                // Attempt to acquire the lock
                if (cpu_lock_state.exchange(1, std::memory_order_acquire) == 0) {
                    return; // Successfully acquired the lock
                }
            }
            // Spin-wait (busy-wait) until the lock is free
        }
    }

    // CPU unlock
    void cpu_unlock() {
        cpu_lock_state.store(0, std::memory_order_release);
    }
};

// TODO implement a MPMC queue
template <typename T, uint8_t size> 
class MPMCqueue
{
private:
    static const size_t N = 1 << size;
    T queue[N];
    cuda::atomic<size_t> _head = 0, _tail = 0;
public:
    __host__ __device__ void push(const T &data) {
        int tail = _tail.load(cuda::memory_order_relaxed);
        while (tail - _head.load(cuda::memory_order_acquire) == N)
            ;
        queue[_tail % N] = data;
        _tail.store(tail + 1, cuda::memory_order_release);
    }

    __host__ __device__ T pop() {
        int head = _head.load(cuda::memory_order_relaxed);
        while (_tail.load(cuda::memory_order_acquire) == _head)
            ;
        T item = queue[_head % N];
        _head.store(head + 1, cuda::memory_order_release);
    return item;
    }
};

// element of data structure
class data_element
{
public:
    int img_id;
    uchar *img_in;
    uchar *img_out;
};

// TODO implement the persistent kernel
__global__
void persistent_kernel(){

}

// TODO implement a function for calculating the threadblocks count
int calculate_threadblocks_count(int threads) {
    // set device
    int device;
    hipDeviceProp_t deviceProp;
    CUDA_CHECK( hipGetDevice(&device) );
    CUDA_CHECK( hipGetDeviceProperties(&deviceProp, device) );
    // get device properties
    int SM_count = deviceProp.multiProcessorCount;
    int max_threads_per_SM = deviceProp.maxThreadsPerMultiProcessor;
    int max_blocks_per_SM = deviceProp.maxBlocksPerMultiProcessor;
    int max_shared_mem_per_SM = deviceProp.sharedMemPerMultiprocessor;
    int max_regs_per_SM = deviceProp.regsPerMultiprocessor;
    // kernel properties
    int threads_per_block = threads;
    int shared_mem_per_block = INTERPOLATE_MEM;
    int regs_per_thread = REGS_PER_THREAD;
    // calculate threadblocks count
    // init with max possible threadblocks count
    int threadblocks = max_blocks_per_SM;
    // check threads constraint per SM
    threadblocks = min(threadblocks, (max_threads_per_SM / threads_per_block));
    // check shared memory constraint per SM
    threadblocks = min(threadblocks, (max_shared_mem_per_SM / shared_mem_per_block));
    // check register constraint per SM
    threadblocks = min(threadblocks, (max_regs_per_SM / (threads_per_block * regs_per_thread)));
    // return threadblocks count per all SMs
    return threadblocks * SM_count;
}

class queue_server : public image_processing_server
{
private:
    // TODO define queue server context (memory buffers, etc...)
    int thread_blocks;
    int max_queue_size;
    // queue for tasks

    // queue for results
public:
    queue_server(int threads)
    {
        // TODO initialize host state
        // TODO launch GPU persistent kernel with given number of threads, and calculated number of threadblocks
        this->thread_blocks = calculate_threadblocks_count(threads);
        this->max_queue_size = 16 * this->thread_blocks;
        // init a queue for tasks and a queue for results

        
    }

    ~queue_server() override
    {
        // TODO free resources allocated in constructor
    }

    bool enqueue(int img_id, uchar *img_in, uchar *img_out) override
    {
        // TODO push new task into queue if possible
        return false;
    }

    bool dequeue(int *img_id) override
    {
        // TODO query (don't block) the producer-consumer queue for any responses.
        return false;

        // TODO return the img_id of the request that was completed.
        //*img_id = ... 
        return true;
    }
};

std::unique_ptr<image_processing_server> create_queues_server(int threads)
{
    return std::make_unique<queue_server>(threads);
}
