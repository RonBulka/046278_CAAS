#include "hip/hip_runtime.h"
#include "ex2.h"
#include <cuda/atomic>
#include <new>

#define STREAM_THREADS 1024
#define COMMON_SIZE 256
#define INTERPOLATE_MEM 1024
#define REGS_PER_THREAD 32

class queue_server;
class streams_server;
class MPMCqueue;

// element of data structure
typedef struct data_element_t {
    int img_id;
    uchar *img_in;
    uchar *img_out;
} data_element;

__global__ void persistent_kernel(uchar* maps, MPMCqueue* tasks,
                                  MPMCqueue* results, cuda::atomic<bool>* stop_kernel);
__device__ void debug_msg(const char* msg, int hist[], int hist_size);
int calculate_threadblocks_count(int threads);
int calculate_upper_log2(int n);

int calculate_upper_log2(int n) {
    int log2 = 0;
    int flag = 0;
    while (n > 1) {
        if (n % 2 == 1) {
            flag = 1;
        }
        n = n >> 1;
        log2++;
    }
    return log2 + flag;
}

__device__ void prefix_sum(int arr[], int arr_size) {
    // TODO complete according to hw1
    int tid = threadIdx.x;
    // int num_threads = blockDim.x;
    // extern __shared__ int temp[];
    int increase;
    for (int stride = 1; stride < arr_size; stride *= 2) {
        // for (int i = tid; i < arr_size; i += num_threads) {
        //     if (i >= stride) {
        //         temp[i] = arr[i - stride];
        //     }
        // }
        if ((tid >= stride) && (tid < arr_size)) {
            increase = arr[tid - stride];
        } 
        __syncthreads();
        // for (int i = tid; i < arr_size; i += num_threads) {
        //     if (i >= stride) {
        //         arr[i] += temp[i];
        //     }
        // }
        if ((tid >= stride) && (tid < arr_size)) {
            arr[tid] += increase;
        }
        __syncthreads();
    }
    return;
}

/**
 * Perform interpolation on a single image
 *
 * @param maps 3D array ([TILES_COUNT][TILES_COUNT][256]) of    
 *             the tiles’ maps, in global memory.
 * @param in_img single input image, in global memory.
 * @param out_img single output buffer, in global memory.
 */
__device__
 void interpolate_device(uchar* maps ,uchar *in_img, uchar* out_img);

__device__
void process_image(uchar *in, uchar *out, uchar* maps) {
    // TODO complete according to hw1
    int tid = threadIdx.x;
    int block_size = blockDim.x;
    __shared__ int sharedHistogram[COMMON_SIZE];
    // do one tile at a time
    for (int row_tile_n = 0; row_tile_n < TILE_COUNT; row_tile_n++) {
        for (int col_tile_n = 0; col_tile_n < TILE_COUNT; col_tile_n++) {
            // Initialize sharedHistogram/reset everything to zero
            for (int k = tid; k < COMMON_SIZE; k += block_size) {
                sharedHistogram[k] = 0;
            }
            __syncthreads();
            
            // // Debug print for shared memory initialization
            // if (tid == 0) {
            //     printf("Thread block %d Processing tile %d %d\n", blockIdx.x, row_tile_n, col_tile_n);
            //     debug_msg("Shared Histogram Initialized:", sharedHistogram, COMMON_SIZE);
            // }
            // __syncthreads();

            // Fill histogram
            for (int i = tid; i < TILE_WIDTH * TILE_WIDTH; i += block_size) {
                int tile_col = i % TILE_WIDTH;
                int tile_row = i / TILE_WIDTH;
                int y = TILE_WIDTH * row_tile_n + tile_row;
                int x = TILE_WIDTH * col_tile_n + tile_col;
                uchar* row = in + y * IMG_WIDTH;
                int pixel = (int)row[x];
                atomicAdd(&sharedHistogram[pixel], 1);
            }
            __syncthreads(); // Ensure all atomic adds are done

            // // Debug print for histogram values
            // if (tid == 0) {
            //     debug_msg("Shared Histogram Filled:", sharedHistogram, COMMON_SIZE);
            // }
            // __syncthreads();

            // Prefix sum on sharedHistogram
            prefix_sum(sharedHistogram, COMMON_SIZE);
            __syncthreads(); // Ensure prefix sum is completed

            // // Debug print for prefix sum values
            // if (tid == 0) {
            //     debug_msg("Shared Histogram After Prefix Sum:", sharedHistogram, COMMON_SIZE);
            // }
            // __syncthreads();

            // Get correct maps entry
            uchar* map = &maps[row_tile_n * TILE_COUNT * COMMON_SIZE + col_tile_n * COMMON_SIZE];
            // Create new map values
            for (int k = tid; k < COMMON_SIZE; k += block_size) {
                map[k] = (float(sharedHistogram[k]) * 255) / (TILE_WIDTH * TILE_WIDTH);
            }
            __syncthreads();
        }
    }
    // if (tid == 0) {
    //     printf("Thread block %d before interpolate_device\n", blockIdx.x);
    // }
    // __syncthreads();
    interpolate_device(maps, in, out);
    return;
}

__device__
void debug_msg(const char* msg, int hist[], int hist_size) {
    printf("%s\n", msg);
    for (int i = 0; i < hist_size; i++) {
        printf("%d: %d\t", i, hist[i]);
    }
    printf("\n");
}

__global__
void process_image_kernel(uchar *in, uchar *out, uchar* maps){
    process_image(in, out, maps);
}

/*****************************************************************************/
// Streams implemintation
/*****************************************************************************/
class streams_server : public image_processing_server
{
private:
    // TODO define stream server context (memory buffers, streams, etc...)
    hipStream_t streams[STREAM_COUNT];
    uchar* maps;
    uchar* in_image;
    uchar* out_image;
    int image_id[STREAM_COUNT];

public:
    streams_server() {
        // TODO initialize context (memory buffers, streams, etc...)
        for (int i = 0; i < STREAM_COUNT; i++) {
            CUDA_CHECK(hipStreamCreate(&streams[i]));
            image_id[i] = -1;
        }
        CUDA_CHECK(hipHostAlloc(&maps, sizeof(uchar) * STREAM_COUNT * TILE_COUNT * TILE_COUNT * COMMON_SIZE, hipHostMallocDefault));
    }

    ~streams_server() override {
        // TODO free resources allocated in constructor
        for (int i = 0; i < STREAM_COUNT; i++) {
            CUDA_CHECK(hipStreamDestroy(streams[i]));
        }
        CUDA_CHECK(hipHostFree(maps));
    }

    bool enqueue(int img_id, uchar *img_in, uchar *img_out) override {
        // TODO place memory transfers and kernel invocation in streams if possible.
        for (int i = 0; i < STREAM_COUNT; i++) {
            if (image_id[i] == -1) {
                image_id[i] = img_id;
                uchar* maps_d = this->maps + i * TILE_COUNT * TILE_COUNT * COMMON_SIZE;
                process_image_kernel<<<1, STREAM_THREADS, 0, streams[i]>>>(img_in, img_out, maps_d);
                return true;
            }
        }
        return false;
    }

    bool dequeue(int *img_id) override {
        // TODO query (don't block) streams for any completed requests.
        for (int i = 0; i < STREAM_COUNT; i++) {
            hipError_t status = hipStreamQuery(streams[i]); // TODO query diffrent stream each iteration
            switch (status) {
            case hipSuccess:
                // TODO return the img_id of the request that was completed.
                if (image_id[i] == -1) {
                    continue;
                }
                *img_id = image_id[i];
                // printf("Image %d is ready\n", *img_id);
                image_id[i] = -1;
                return true;
            case hipErrorNotReady:
                return false;
            default:
                CUDA_CHECK(status);
                return false;
            }
        }
        return false;
    }
};

std::unique_ptr<image_processing_server> create_streams_server()
{
    return std::make_unique<streams_server>();
}

/*****************************************************************************/
// Queue implemintation
/*****************************************************************************/
// TODO implement a Test and Test and Set lock
class TATASLock {
private:
    cuda::atomic<int, cuda::thread_scope_device>* gpu_lock_state;

public:
    __host__ TATASLock() {
        // Initialize lock state
        CUDA_CHECK(hipMalloc(&gpu_lock_state, sizeof(cuda::atomic<int, cuda::thread_scope_device>)));
        CUDA_CHECK(hipMemset(gpu_lock_state, 0, sizeof(cuda::atomic<int, cuda::thread_scope_device>)));
    }

    __host__ ~TATASLock() {
        // Free resources allocated in constructor
        CUDA_CHECK(hipFree(gpu_lock_state));
    }

    __device__ void lock() {
        while (true) {
            if (gpu_lock_state->load(cuda::memory_order_acquire) == 0) {
                if (gpu_lock_state->exchange(1, cuda::memory_order_acquire) == 0) {
                    return;
                }
            }
        }
    }

    __device__ void unlock() {
        gpu_lock_state->store(0, cuda::memory_order_release);
    }
};

class MPMCqueue {
private:
    size_t max_size;
    data_element* queue;
    cuda::atomic<size_t>* _head;
    cuda::atomic<size_t>* _tail;
    TATASLock lock;

public:
    __host__ MPMCqueue(size_t N) : max_size(N) {
        // Allocate memory for queue
        CUDA_CHECK( hipHostAlloc(&(this->queue), sizeof(data_element) * this->max_size, hipHostMallocDefault));
        CUDA_CHECK( hipHostAlloc(&this->_head, sizeof(cuda::atomic<size_t>), hipHostMallocDefault) );
        ::new(this->_head) cuda::atomic<size_t>(0);
        CUDA_CHECK( hipHostAlloc(&this->_tail, sizeof(cuda::atomic<size_t>), hipHostMallocDefault) );
        ::new(this->_tail) cuda::atomic<size_t>(0);
    }

    __host__ ~MPMCqueue() {
        // Free resources allocated in constructor
        if (this->queue != nullptr) {
            CUDA_CHECK(hipHostFree(this->queue));
        }
        if (this->_head != nullptr) {
            this->_head->~atomic<size_t>();
            CUDA_CHECK(hipHostFree(this->_head));
        }
        if (this->_tail != nullptr) {
            this->_tail->~atomic<size_t>();
            CUDA_CHECK(hipHostFree(this->_tail));
        }
    }

    __device__ bool gpu_push(const data_element &item) {
        lock.lock();
        size_t tail = _tail->load(cuda::memory_order_relaxed);
        if (tail - _head->load(cuda::memory_order_acquire) == max_size) {
            lock.unlock();
            return false;
        }
        queue[tail % max_size] = item;
        _tail->store(tail + 1, cuda::memory_order_release);
        lock.unlock();
        // printf("Thread block %d pushed image %d\n", blockIdx.x, item.img_id);
        return true;
    }

    __device__ bool gpu_pop(data_element *item) {
        lock.lock();
        size_t head = _head->load(cuda::memory_order_relaxed);
        if (_tail->load(cuda::memory_order_acquire) == head) {
            lock.unlock();
            return false;
        }
        *item = queue[head % max_size];
        _head->store(head + 1, cuda::memory_order_release);
        lock.unlock();
        // printf("Thread block %d popped image %d with value of %d in last place\n", blockIdx.x, item->img_id, item->img_in[IMG_HEIGHT * IMG_WIDTH - 1]);
        return true;
    }

    __device__ bool is_empty_gpu() {
        return (_head->load(cuda::memory_order_relaxed) == _tail->load(cuda::memory_order_relaxed));
    }

    __host__ bool cpu_push(const data_element &item) {
        size_t tail = _tail->load(cuda::memory_order_relaxed);
        if (tail - _head->load(cuda::memory_order_acquire) == max_size) {
            return false;
        }
        queue[tail % max_size] = item;
        _tail->store(tail + 1, cuda::memory_order_release);
        return true;
    }

    __host__ bool cpu_pop(data_element *item) {
        size_t head = _head->load(cuda::memory_order_relaxed);
        if (_tail->load(cuda::memory_order_acquire) == head) {
            return false;
        }
        *item = queue[head % max_size];
        _head->store(head + 1, cuda::memory_order_release);
        return true;
    }

    __host__ bool is_empty_cpu() {
        return (_head->load(cuda::memory_order_relaxed) == _tail->load(cuda::memory_order_relaxed));
    }

    __host__ bool is_full_cpu() {
        return (_tail->load(cuda::memory_order_relaxed) - _head->load(cuda::memory_order_relaxed) == max_size);
    }
};

int calculate_threadblocks_count(int threads) {
    // get device properties
    int device;
    hipDeviceProp_t deviceProp;
    CUDA_CHECK(hipGetDevice(&device));
    CUDA_CHECK(hipGetDeviceProperties(&deviceProp, device));
    int SM_count = deviceProp.multiProcessorCount;
    int max_threads_per_SM = deviceProp.maxThreadsPerMultiProcessor;
    int max_blocks_per_SM = deviceProp.maxBlocksPerMultiProcessor;
    int max_shared_mem_per_SM = deviceProp.sharedMemPerMultiprocessor;
    int max_regs_per_SM = deviceProp.regsPerMultiprocessor;

    // get block properties
    int threads_per_block = threads;
    int shared_mem_per_block = INTERPOLATE_MEM + sizeof(int) * COMMON_SIZE + sizeof(data_element) + sizeof(bool);
    int regs_per_thread = REGS_PER_THREAD;

    // calculate threadblocks
    int threadblocks = max_blocks_per_SM;
    // thread constraint
    threadblocks = min(threadblocks, (max_threads_per_SM / threads_per_block));
    // shared memory constraint
    threadblocks = min(threadblocks, (max_shared_mem_per_SM / shared_mem_per_block));
    // register constraint
    threadblocks = min(threadblocks, (max_regs_per_SM / (threads_per_block * regs_per_thread)));

    return threadblocks * SM_count;
}

__global__
void persistent_kernel(uchar* maps, MPMCqueue* tasks, MPMCqueue* results, cuda::atomic<bool>* stop_kernel) {
    // if (threadIdx.x == 0) {
    //     printf("Thread block %d is alive\n", blockIdx.x);
    // }
    __shared__ bool flag;
    __shared__ data_element task;
    uchar* block_maps = maps + blockIdx.x * TILE_COUNT * TILE_COUNT * COMMON_SIZE;
    while (true) {
        if (threadIdx.x == 0) {
            flag = stop_kernel->load(cuda::memory_order_seq_cst) && tasks->is_empty_gpu();
        }
        __syncthreads();
        if (flag) {
            break;
        }
        if (threadIdx.x == 0) {
            flag = tasks->gpu_pop(&task);
        }
        __syncthreads();
        if (!flag) {
            continue;
        }
        __syncthreads();
        process_image(task.img_in, task.img_out, block_maps);
        __syncthreads();
        if (threadIdx.x == 0) {
            // printf("Thread block %d finished processing image %d\n", blockIdx.x, task.img_id);
            while(!results->gpu_push(task));
        }
        __syncthreads();
    }
}

class queue_server : public image_processing_server {
private:
    int thread_blocks;
    int max_queue_size;
    uchar* pinned_queues;
    MPMCqueue* tasks;
    MPMCqueue* results;
    cuda::atomic<bool>* stop_kernel;
    uchar* taskmaps;

public:
    queue_server(int threads) {
        // Allocate memory for atomic<bool>
        CUDA_CHECK( hipHostAlloc(&this->stop_kernel, sizeof(cuda::atomic<bool>), hipHostMallocDefault) );
        ::new(this->stop_kernel) cuda::atomic<bool>(false);

        thread_blocks = calculate_threadblocks_count(threads);
        // printf("Thread blocks: %d\n", thread_blocks);
        max_queue_size = 1 << calculate_upper_log2(thread_blocks << 4);
        // printf("Max queue size: %d\n", max_queue_size);

        // Allocate memory for queues
        CUDA_CHECK(hipHostMalloc(&pinned_queues, sizeof(MPMCqueue) * 2));
        tasks   = new (pinned_queues) MPMCqueue(max_queue_size);
        results = new (pinned_queues + sizeof(MPMCqueue)) MPMCqueue(max_queue_size);
        CUDA_CHECK(hipMalloc(&taskmaps, thread_blocks * TILE_COUNT * TILE_COUNT * COMMON_SIZE * sizeof(uchar)));

        dim3 _threads(threads), blocks(thread_blocks);
        persistent_kernel<<<blocks, _threads>>>(taskmaps, tasks, results, stop_kernel);
    }

    ~queue_server() override {
        // Send signal to stop kernel and wait for it to finish
        stop_kernel->store(true, cuda::memory_order_seq_cst);
        hipDeviceSynchronize();
        hipError_t error = hipGetLastError();
        if (error != hipSuccess) {
            fprintf(stderr, "Kernel execution failed: %s\n", hipGetErrorString(error));
            return;
        }
        // Free resources allocated in constructor
        if (stop_kernel != nullptr) {
            stop_kernel->~atomic<bool>();
            CUDA_CHECK(hipHostFree(stop_kernel));
        }
        // printf("Freeing resources\n");
        if (tasks != nullptr) {
            this->tasks->~MPMCqueue();
        }
        if (results != nullptr) {
            this->results->~MPMCqueue();
        }
        if (pinned_queues != nullptr) {
            CUDA_CHECK(hipHostFree(pinned_queues));
        }
        CUDA_CHECK(hipFree(taskmaps));
    }

    bool enqueue(int img_id, uchar* img_in, uchar* img_out) override {
        data_element task = {};
        task.img_id = img_id;
        task.img_in = img_in;
        task.img_out = img_out;
        return tasks->cpu_push(task);
    }

    bool dequeue(int* img_id) override {
        data_element task;
        if (!results->cpu_pop(&task)) {
            return false;
        }
        *img_id = task.img_id;
        // printf("Image %d is ready\n", *img_id);
        return true;
    }
};

std::unique_ptr<image_processing_server> create_queues_server(int threads)
{
    return std::make_unique<queue_server>(threads);
}